#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>

__global__ void cuda_hello(){
    int i = 0;
    
    printf("hello world from kernel %d :)\n", threadIdx);
    
}

int main() {
    cuda_hello<<<1,32>>>(); 

    hipDeviceSynchronize();

    hipDeviceReset();
    return 0;
}